#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <chrono>
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>


#define att_mask 0b111110
#define left_mask 0b11111111111000000
#define right_mask 0b1111111111100000000000000000
#define class_mask 0b11110000000000000000000000000000
#define leaf_mask 0b1

#define left_shift 6
#define right_shift 17
#define class_shift 28


#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

void importTable(char *path, int *table)
{
    FILE *fp;
    char *line = NULL;
    size_t len = 0;
    ssize_t read;
    int i =0;

    // printf("Import Tables\n");


    fp = fopen(path, "r");
        if (fp == NULL){
            printf("IMPORT ERRO\n");
            exit(EXIT_FAILURE);

        }

    while ((read = getline(&line, &len, fp)) != -1) {
            // printf("%d\n", atoi(line));
            table[i] = atoi(line);
            i++;
        }

    free(line);

}

void importTH(char *path, float *table)
{
    FILE *fp;
    char *line = NULL;
    size_t len = 0;
    ssize_t read;
    int i =0;


    // printf("Import TH/values\n");


    fp = fopen(path, "r");
        if (fp == NULL){
            printf("IMPORT ERRO\n");
            exit(EXIT_FAILURE);

        }

    while ((read = getline(&line, &len, fp)) != -1) {
            // printf("%f\n", atof(line));
            table[i] = atof(line);
            i++;
        }

    free(line);




}

void printVector(int *vector, int size)
{
    for(int i=0;i<size;i++)
        printf("%d : %d\n",i,vector[i]);
}

void printVectorF(float *vector, int size)
{
    for(int i=0;i<size;i++)
        printf("%d : %.2f\n",i,vector[i]);
}



/*
 * Function:  initialData 
 * --------------------
 * initialize all data using a src as source
 *
 *  dest: vector that will store all final data
 *  src: vecto with data that will be copy to dest
 *  items_n: size of src
 *  copies_n: number of copies of src that will be store in dest
 *
 *  returns: the vector dest filled with copies_n of src
 */
void initialData(float*        dest, 
                  const float*  src, 
                  size_t        items_n,
                  size_t        copies_n)
{
  for(size_t i=0; i<copies_n; i++)
  {
    memcpy(&dest[i * items_n], 
           src, 
           sizeof(*src) * items_n);
  }

    // printf("Inital Data\n");

  
}

void checkResult(int *P,int *proof, int nElem, int nProof){
    
    for(int i=0;i<nElem;i+=nProof){
        for(int j=0;j<nProof;j++){
            assert(P[i+j] == proof[j]);
        }
    }

}

__global__ void table_RF(float *att_table, float *values_table, int *tree_table, int *P, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int next = 0;
    int atr = 0;
    int left = 0;
    int right = 0;
    float th = 0;

    if(idx < N)
    {

        while((tree_table[next] & leaf_mask) == 0)
        {
            atr = (tree_table[next] & att_mask) >> 1;
            left = (tree_table[next] & left_mask) >> left_shift;
            right = (tree_table[next] & right_mask) >> right_shift;
            th = att_table[next];
            next = (values_table[(4*idx)+atr] > th ? right:left);


        }

    
        P[idx] = (tree_table[next] & class_mask) >> class_shift;

    }



}


int main()
{

    float elapsed_time;
    char path[] = "/home/olavo/random_forest_acelerator/misc/bench/tree0/tree.txt";
    char path_th[] = "/home/olavo/random_forest_acelerator/misc/bench/tree0/th.txt";
    char path_values[] = "/home/olavo/random_forest_acelerator/misc/bench/tree0/values.txt";

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    int nElem = 1 << 20;
    int nThreads = 7;
    int nBlocks = 1;
    int nBytes = nElem * sizeof(float);
    int *tree_table,*P;
    float *att_table,*values,*values_copy;

    att_table     = (float *)malloc(nBytes);
    values     = (float *)malloc(nBytes);
    values_copy     = (float *)malloc(28 * sizeof(float));
    tree_table  = (int *)malloc(nBytes);
    P  = (int *)malloc(nBytes);

    printf("Number of elements: %d\n",nElem);

    importTable(path,tree_table);
    importTH(path_th,att_table);
    importTH(path_values,values_copy);

    initialData(values,values_copy,28,nElem/28);

    

    float *d_values, *d_att;
    int *d_table,*d_P;
    CHECK(hipMalloc((float**)&d_values, nBytes));
    CHECK(hipMalloc((float**)&d_att, nBytes));
    CHECK(hipMalloc((int**)&d_P, nBytes));
    CHECK(hipMalloc((int**)&d_table, nBytes));



    // transfer data from host to device
    CHECK(hipMemcpy(d_att, att_table, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_values, values, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_table, tree_table, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int iLen = 512;
    dim3 block (iLen);
    dim3 grid  ((nElem + block.x - 1) / block.x);

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

     //Código GPU
    // record start event
    CHECK(hipEventRecord(start, 0));   
    table_RF<<<nElem/nThreads, nBlocks>>>(d_att, d_values, d_table,d_P,nElem);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("RF with TABLE - execution time = %.6fms\n",
           elapsed_time );

    CHECK(hipGetLastError());
    
    // copy kernel result back to host side
    CHECK(hipMemcpy(P, d_P, nBytes, hipMemcpyDeviceToHost));


    /* Check results*/
    int proof[7] = {0,1,2,1,2,1,2};
    checkResult(P,proof,((nElem/nThreads)-3),7);

    /* Free all memory*/
    CHECK(hipFree(d_att));
    CHECK(hipFree(d_values));
    CHECK(hipFree(d_table));
    CHECK(hipFree(d_P));

    free(tree_table);
    free(values);
    free(values_copy);
    free(att_table);
    free(P);




    return 0;
    
}